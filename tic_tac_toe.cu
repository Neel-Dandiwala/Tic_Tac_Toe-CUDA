/**
*   Author: Neel Dandiwala
*   Date:   October 2022
**/

#include "hip/hip_runtime.h"


#include <hip/hip_runtime_api.h>
#include <iostream>
#include <stdio.h>

#define WIDTH 3

__constant__ char player;
__constant__ char opponent;


#define cudaCHECK(res) { cudaASSERT((res), __FILE__, __LINE__);}
inline void cudaASSERT(hipError_t err, const char *file, int line, bool abort=true){
    if(err != hipSuccess) {
        fprintf(stderr, "GPUassert: %s %s %d\n", hipGetErrorString(err), file, line);
        if (abort) exit(err);
    }
}

__device__ __host__ void display_board(char *d_board){
    printf("\n\nPlayer 1 [X] \t Player 2 [O]\n\n");
    printf("\t\t       |       |       \n");
    printf("\t\t   %c   |   %c   |   %c\n", d_board[0], d_board[1], d_board[2]);
    printf("\t\t_______|_______|_______\n");
    printf("\t\t       |       |       \n");
    printf("\t\t   %c   |   %c   |   %c\n", d_board[3], d_board[4], d_board[5]);
    printf("\t\t_______|_______|_______\n");
    printf("\t\t       |       |       \n");
    printf("\t\t   %c   |   %c   |   %c\n", d_board[6], d_board[7], d_board[8]);
    printf("\t\t       |       |       \n");
}

__device__ bool emptySlots(char *d_board){
    for(int i = 0; i < WIDTH * WIDTH; i++){
        if(d_board[i] == '_') return true;
    }

    return false;
}

__device__ bool validation(char *d_board){

    // Victories on Rows
    if(d_board[0] == d_board[1] && d_board[1] == d_board[2]){
        if(d_board[0] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }
    if(d_board[3] == d_board[4] && d_board[4] == d_board[5]){
        if(d_board[3] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }
    if(d_board[6] == d_board[7] && d_board[7] == d_board[8]){
        if(d_board[6] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }

    // Victories on Columns
    if(d_board[0] == d_board[3] && d_board[3] == d_board[6]){
        if(d_board[0] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }
    if(d_board[1] == d_board[4] && d_board[4] == d_board[7]){
        if(d_board[1] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }
    if(d_board[2] == d_board[5] && d_board[5] == d_board[8]){
        if(d_board[2] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }

    // Victories on Diagonals
    if(d_board[0] == d_board[4] && d_board[4] == d_board[8]){
        if(d_board[4] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }
    if(d_board[2] == d_board[4] && d_board[4] == d_board[6]){
        if(d_board[2] == player){
            display_board(d_board);
            printf("Player 1 [X] wins! \n\n");
            return false;
        } else {
            display_board(d_board);
            printf("Player 2 [O] wins! \n\n");
            return false;
        }
    }

    return true;
}

__device__ int score(char *d_board){

    // Victories on Rows
    if(d_board[0] == d_board[1] && d_board[1] == d_board[2]){
        if(d_board[0] == player){
            return 10;
        } else {
            return -10;
        }
    }
    if(d_board[3] == d_board[4] && d_board[4] == d_board[5]){
        if(d_board[3] == player){
            return 10;
        } else {
            return -10;
        }
    }
    if(d_board[6] == d_board[7] && d_board[7] == d_board[8]){
        if(d_board[6] == player){
            return 10;
        } else {
            return -10;
        }
    }

    // Victories on Columns
    if(d_board[0] == d_board[3] && d_board[3] == d_board[6]){
        if(d_board[0] == player){
            return 10;
        } else {
            return -10;
        }
    }
    if(d_board[1] == d_board[4] && d_board[4] == d_board[7]){
        if(d_board[1] == player){
            return 10;
        } else {
            return -10;
        }
    }
    if(d_board[2] == d_board[5] && d_board[5] == d_board[8]){
        if(d_board[2] == player){
            return 10;
        } else {
            return -10;
        }
    }

    // Victories on Diagonals
    if(d_board[0] == d_board[4] && d_board[4] == d_board[8]){
        if(d_board[4] == player){
            return 10;
        } else {
            return -10;
        }
    }
    if(d_board[2] == d_board[4] && d_board[4] == d_board[6]){
        if(d_board[2] == player){
            return 10;
        } else {
            return -10;
        }
    }

    return 0;
}

__device__ int minmax(char *d_board, int depth, bool isMax){
    int benefit = score(d_board);

    if(benefit == 10) return (benefit - depth);

    if(benefit == -10) return (depth - 10);

    if(emptySlots(d_board) == false) return 0;

    if(isMax){

        int maximumBenefit = -1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){
            if(d_board[i] == '_'){

                d_board[i] = player;

                int tempBenefit = minmax(d_board, depth + 1, !isMax);

                d_board[i] = '_';

                maximumBenefit = tempBenefit > maximumBenefit ? tempBenefit : maximumBenefit;
            }
        }

        return maximumBenefit;

    } else {
        
        int minimumBenefit = 1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){
            if(d_board[i] == '_'){

                d_board[i] = opponent;

                int tempBenefit = minmax(d_board, depth + 1, !isMax);

                d_board[i] = '_';

                minimumBenefit = minimumBenefit > tempBenefit ? tempBenefit : minimumBenefit;

            }
        }

        return minimumBenefit;

    }
}

__device__ void executeMove(char *d_board, bool isPlayer, int finalPosition){
    if(isPlayer) {
        d_board[finalPosition] = player;
    } else {
        d_board[finalPosition] = opponent;
    }
    display_board(d_board);
}

__device__ int nextMove(char *d_board, bool isPlayer){
    
    int initialBenefit;
    int position = -1;
    if(isPlayer){
        initialBenefit = -1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){
            
            if(d_board[i] == '_'){

                d_board[i] = player;
                int tempBenefit = minmax(d_board, 0, false);
                d_board[i] = '_';

                if(initialBenefit < tempBenefit){
                    initialBenefit = tempBenefit;
                    position = i;
                }

            }

        }

    } else {
        initialBenefit = 1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){

            if(d_board[i] == '_'){

                d_board[i] = opponent;
                int tempBenefit = minmax(d_board, 0, true);
                d_board[i] = '_';

                if(initialBenefit > tempBenefit){
                    initialBenefit = tempBenefit;
                    position = i;
                }

            }

        }

    }

    return position;

}

// The MinMax implementation
__global__ void ticTacToe(char *d_board, int *d_isPlayer){

    // display_board(d_board);
    printf("Im here\n");
    printf("TRUE: %d\n", *d_isPlayer);
    printf("Player %c : Opponent %c \n", player, opponent);
    bool isPlayer = true;
    while(validation(d_board) && emptySlots(d_board)){
        int finalPosition = nextMove(d_board, isPlayer);
        executeMove(d_board, isPlayer, finalPosition);
        isPlayer = !isPlayer;
    }

}




int main(void) {

    printf("Tic Tac Toe\n");

    char playerSymbol = 'X';
    char opponentSymbol = 'O';

    cudaCHECK(hipMemcpyToSymbol(HIP_SYMBOL(player), &playerSymbol, sizeof(char)));
    cudaCHECK(hipMemcpyToSymbol(HIP_SYMBOL(opponent), &opponentSymbol, sizeof(char)));

    const unsigned int size = WIDTH * WIDTH;

    char h_board[size] = {
        '_', '_', '_',
        '_', '_', '_',
        '_', '_', '_'
    };

    char *d_board;

    cudaCHECK(hipMalloc((void**)&d_board, size * sizeof(char)));
    cudaCHECK(hipMemcpy(d_board, h_board, size * sizeof(char), hipMemcpyHostToDevice));

    int h_isPlayer = 1;
    int *d_isPlayer;

    cudaCHECK(hipMalloc(&d_isPlayer, sizeof(int)));
    cudaCHECK(hipMemcpy(d_isPlayer, &h_isPlayer, sizeof(int),hipMemcpyHostToDevice));

    printf("Done\n");

    std::cout << h_isPlayer << std::endl;

    ticTacToe<<<1,1>>>(d_board, d_isPlayer);
    hipDeviceSynchronize();

    cudaCHECK(hipMemcpy(h_board, d_board, size * sizeof(char), hipMemcpyDeviceToHost));
    display_board(h_board);
    return 0;
}

