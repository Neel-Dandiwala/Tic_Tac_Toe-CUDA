/**
* Author: Neel Dandiwala
* Date: October 2022
**/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <array>
using namespace std;

#define WIDTH 3

char board[WIDTH * WIDTH] = {
    '_', '_', '_',
    '_', '_', '_',
    '_', '_', '_'
};

char player = 'X';
char opponent = 'O';

int row1[3] = {0, 1, 2};
int row2[3] = {3, 4, 5};
int row3[3] = {6, 7, 8};
int col1[3] = {0, 3, 6};
int col2[3] = {1, 4, 7};
int col3[3] = {2, 5, 8};
int dia1[3] = {0, 4, 8};
int dia2[3] = {2, 4, 6};
int* winningsets[8] = {row1, row2, row3, col1, col2, col3, dia1, dia2};

bool emptySlots(char *board){
    for(int i = 0; i < WIDTH * WIDTH; i++){
        if(board[i] == '_') return true;
    }

    return false;
}

void display_board(){
    cout << "\n\nPlayer 1 [X] \t Player 2 [O]\n\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[0] <<"   |   " << board[1] << "   |   " << board[2] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[3] <<"   |   " << board[4] << "   |   " << board[5] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[6] <<"   |   " << board[7] << "   |   " << board[8] << endl;
    cout << "\t\t       |       |       \n";
}

bool validation(char *board){
    for(auto v : winningsets){
        if(board[v[0]] == board[v[1]] && board[v[1]] == board[v[2]]){
            if(board[v[0]] == player) {
                display_board();
                cout << "Player 1 [X] wins! \n\n";
                return false;
            } else if(board[v[0]] == opponent){
                display_board();
                cout << "Player 2 [O] wins! \n\n";
                return false;
            }
        }
    } 

    return true;
}

int score(char *board){
    for(auto v : winningsets){
        if(board[v[0]] == board[v[1]] && board[v[1]] == board[v[2]]){
            if(board[v[0]] == player) {
                return 10;
            } else if(board[v[0]] == opponent){
                return -10;
            }
        }
    } 

    return 0;
}

int minmax(char *board, int depth, bool isMax){

    int benefit = score(board);

    if(benefit == 10) return (benefit - depth);

    if(benefit == -10) return (depth - 10);

    if(emptySlots(board) == false) return 0;

    if(isMax){

        int maximumBenefit = -1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){
            if(board[i] == '_'){

                board[i] = player;

                int tempBenefit = minmax(board, depth + 1, !isMax);

                board[i] = '_';

                maximumBenefit = max(maximumBenefit, tempBenefit);

            }
        }

        return maximumBenefit;
    } else {

        int minimumBenefit = 1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){

            if(board[i] == '_'){

                board[i] = opponent;

                int tempBenefit = minmax(board, depth + 1, !isMax);

                board[i] = '_';

                minimumBenefit = min(minimumBenefit, tempBenefit);

            }
        }

        return minimumBenefit;
    }
}

int nextMove(char *board, bool isPlayer){

    int initialBenefit;
    int position = -1;
    if(isPlayer){
        initialBenefit = -1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){

            if(board[i] == '_'){

                board[i] = player;
                int tempBenefit = minmax(board, 0, false);
        
                board[i] = '_';

                if(initialBenefit < tempBenefit){
                    initialBenefit = tempBenefit;
                    position = i;
                }

            }
        }

        // return position;
    } else {
        initialBenefit = 1000;

        for(int i = 0; i < WIDTH * WIDTH; i++){

            if(board[i] == '_'){

                board[i] = opponent;
                int tempBenefit = minmax(board, 0, true);
        
                board[i] = '_';

                if(initialBenefit > tempBenefit){
                    initialBenefit = tempBenefit;
                    position = i;
                }

            }
        }

        // return position;
    }
    

    return position;
}

void executeMove(char *board, bool isPlayer, int finalPosition){
    if(isPlayer) {
        board[finalPosition] = player;
    } else {
        board[finalPosition] = opponent;
    }
    display_board();
}

int main(){
    display_board();
    bool isPlayer = true;
    while(validation(board) && emptySlots(board)){
        int finalPosition = nextMove(board, isPlayer);
        executeMove(board, isPlayer, finalPosition);
        isPlayer = !isPlayer;
    }

}
