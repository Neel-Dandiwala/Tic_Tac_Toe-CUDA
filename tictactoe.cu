/**
*
*
*
**/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
using namespace std;

int hor1[3] = {0, 1, 2};
int hor2[3] = {3, 4, 5};
int hor3[3] = {6, 7, 8};
int ver1[3] = {0, 3, 6};
int ver2[3] = {1, 4, 7};
int ver3[3] = {2, 5, 8};
int dia1[3] = {0, 4, 8};
int dia2[3] = {2, 4, 6};
int* winningsets[8] = {hor1, hor2, hor3, ver1, ver2, ver3, dia1, dia2};


char board[3][3] = {
    {'1', '2', '3'},
    {'4', '5', '6'},
    {'7', '8', '9'}
};

int choice;
int row, column;

int player1[9] = {0};
int player2[9] = {0};

char turn = 'X';

bool game = true;
bool draw = false;

void display_board(){
    cout << "\n\nPlayer 1 [X] \t Player 2 [O]\n\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[0][0] <<"   |   " << board[0][1] << "   |   " << board[0][2] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[1][0] <<"   |   " << board[1][1] << "   |   " << board[1][2] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[2][0] <<"   |   " << board[2][1] << "   |   " << board[2][2] << endl;
    cout << "\t\t       |       |       \n";
}

void selection(){

    cout << "\n Enter your choice: \n";

    cin >> choice;

    switch(choice) {
        case 1: 
            row = 0;
            column = 0;
            break;
        case 2: 
            row = 0;
            column = 1;
            break;
        case 3: 
            row = 0;
            column = 2;
            break;
        case 4: 
            row = 1;
            column = 0;
            break;
        case 5: 
            row = 1;
            column = 1;
            break;
        case 6: 
            row = 1;
            column = 2;
            break;
        case 7: 
            row = 2;
            column = 0;
            break;
        case 8: 
            row = 2;
            column = 1;
            break;
        case 9: 
            row = 2;
            column = 2;
            break;
        default:
            cout << "\n Invalid Move \n";
    }

}

void validation(int *player, int num){
    cout << "\n Validation \n"; 
    for(auto v : winningsets){
        if(player[v[0]] == 1 && player[v[1]] == 1 && player[v[2]] == 1) {
            cout << "\n PLAYER " << num << " WONNNN \n";
            game = false;
            display_board();
        }
    }

}

void player_turn(){
    if (turn == 'X'){
        cout << "\n Player 1 [X] play : \n";
    }
    else if (turn == 'O'){
        cout << "\n PLayer 2 [O] play:  \n";
    }

    selection();
    if(turn == 'X' && board[row][column] != 'X' && board[row][column] != 'O'){
        board[row][column] = 'X';
        player1[choice - 1] = 1;
        for (int i = 0; i < sizeof(player1) / sizeof(int); i++){
            cout << player1[i] << "   ";
        }
        validation(player1, 1);
        turn = 'O';
    } else if (turn == 'O' && board[row][column] != 'X' && board[row][column] != 'O'){
        board[row][column] = 'O';
        player2[choice - 1] = 1;
        for (int i = 0; i < sizeof(player2) / sizeof(int); i++){
            cout << player2[i] << "   ";
        }
        validation(player2, 2);
        turn = 'X';
    } else {
        cout << "\n Slot is already filled! Invalid Move! Try again\n";
        selection();
    }
}


int main(){
    cout << "\t\t\t TIC TAC TOE \t\t\t";
    
for(int i = 0; i < 8; i++){
    cout << winningsets[i][1] <<"\n ";
}


    while(game == true){
        display_board();
        player_turn();
    }
}