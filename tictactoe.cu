/**
* Author: Neel Dandiwala
* Date: September 2022
*
**/


#include <hip/hip_runtime.h>
#include <iostream>
#include <stdlib.h>
#include <bits/stdc++.h>
#include <array>
using namespace std;

const int width = 3;

int hor1[3] = {0, 1, 2};
int hor2[3] = {3, 4, 5};
int hor3[3] = {6, 7, 8};
int ver1[3] = {0, 3, 6};
int ver2[3] = {1, 4, 7};
int ver3[3] = {2, 5, 8};
int dia1[3] = {0, 4, 8};
int dia2[3] = {2, 4, 6};
int* winningsets[8] = {hor1, hor2, hor3, ver1, ver2, ver3, dia1, dia2};


char board[width][width] = {
    {'1', '2', '3'},
    {'4', '5', '6'},
    {'7', '8', '9'}
};

// int slots[width * width] = {0};
int slots[width * width] = {1, 0, 0, 0, 0, 0, 0, 0, 0};
int player1[width * width] = {1, 0, 0, 0, 0, 0, 0, 0, 0};

int choice;
int row, column;

// int player1[width * width] = {0};
int player2[width * width] = {0};

int turn = 1;

bool winning_move = false;
bool game = true;
bool draw = false;

void copy_array(int *temp, int *original){
    for(int i = 0; i < width * width; i++){
        temp[i] = original[i];
    }
}

void display_board(){
    cout << "\n\nPlayer 1 [X] \t Player 2 [O]\n\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[0][0] <<"   |   " << board[0][1] << "   |   " << board[0][2] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[1][0] <<"   |   " << board[1][1] << "   |   " << board[1][2] << endl;
    cout << "\t\t_______|_______|_______\n";
    cout << "\t\t       |       |       \n";
    cout << "\t\t   " << board[2][0] <<"   |   " << board[2][1] << "   |   " << board[2][2] << endl;
    cout << "\t\t       |       |       \n";
}


void selection(){
    cout << "\n The choice: " << choice << "\n";

    switch(choice) {
        case 0: 
            row = 0;
            column = 0;
            break;
        case 1: 
            row = 0;
            column = 1;
            break;
        case 2: 
            row = 0;
            column = 2;
            break;
        case 3: 
            row = 1;
            column = 0;
            break;
        case 4: 
            row = 1;
            column = 1;
            break;
        case 5: 
            row = 1;
            column = 2;
            break;
        case 6: 
            row = 2;
            column = 0;
            break;
        case 7: 
            row = 2;
            column = 1;
            break;
        case 8: 
            row = 2;
            column = 2;
            break;
        default:
            cout << "\n Invalid Move \n";
            break;
    }

}

bool validation(int *player, bool winning_move){
    // cout << "\n Validation \n"; 

    for(auto v : winningsets){
        if(player[v[0]] == 1 && player[v[1]] == 1 && player[v[2]] == 1) {
            if(winning_move)
            {
                cout << "\n PLAYER " << " WONNNN \n";
                game = false;
                display_board();
            }
            return true;
        }
    }

    winning_move = false;
    return false;

}


int score(int depth, int* player, int* opponent) {
    if(validation(player, false)){
        return 10 - depth;
    } else if(validation(opponent, false)) {
        return depth - 10;
    }

    return 0;
}

int check_scenario(int i, int depth, int* given_slots, int* player, int* opponent){
        given_slots[i] = 1;
        player[i] = 1;
        int max_score = INT_MIN;
        int temp;
        //bool toggle_turn = true;
        if(score(depth, player, opponent) == 0){
            depth += 1;
            set<int> s;
            set<int>::iterator it;
            for(int i = 0; i < width * width; i++){
                if(given_slots[i] == 0){
                    given_slots[i] = 1;
                    opponent[i] = 1;  
                    for(int j = 0; j < width * width; j++){
                        if (given_slots[j] == 0){
                            temp = check_scenario(j, depth, given_slots, player, opponent); 
                            s.insert(temp);
                        }
                    } 
                    cout << "\n TEMPPP: " << temp << "\n";

                }

            }

            for(auto it : s){
                if(it >= max_score){
                    max_score = it;
                }
            }
            return max_score;
        }
        return score(depth, player, opponent);
   
}

void check_slot(int* given_slots, int* player, char sym){
    if(given_slots[width * row + column] == 0){
        board[row][column] = sym;
        given_slots[width * row + column] = 1;
        player[width * row + column] = 1;
        validation(player, true);
        turn = !turn;
        return;
    } else {
        cout << "\n Slot is already filled! Invalid Move! Try again\n";
        turn = 1;
    }
}

int scenario(int *current_slots, int *player, int* opponent){
    int sample_slots[width * width];
    copy_array(sample_slots, current_slots);
    int sample_player[width * width];
    copy_array(sample_player, player);
    int sample_opponent[width * width];
    copy_array(sample_opponent, opponent);

    int position;
    int max_score = INT_MIN;
    int temp;
    map<int, int> m;
    map<int, int>::iterator iter;
    int depth = 0;
    int index = 0;
    int threshold = (width * width);
    while(index < threshold){
        if(current_slots[index] == 0){
            temp = check_scenario(index, depth, sample_slots, sample_player, sample_opponent);
            m.insert(pair<int, int>(index, temp));
        } 
        index += 1;
    }


    for(iter=m.begin(); iter!=m.end(); iter++){
        if(iter->second >= max_score){
            
            max_score = iter->second;
            position = iter->first;
        }
    }

    cout << "\nMAX SCORE: " << max_score <<" & POSITION: " << position <<endl;

    return position;
}

void player_turn(){
    if (turn == 1){
        cout << "\n Player 1 [X] play : \n";
        choice = scenario(slots, player1, player2);
        selection();
        check_slot(slots, player1, 'X');
    }
    else if (turn == 0){
        cout << "\n PLayer 2 [O] play:  \n";
        choice = scenario(slots, player2, player1);
        selection();
        check_slot(slots, player2, 'O');
            
    }
}



int main(){
    cout << "\t\t\t TIC TAC TOE \t\t\t";

    while(game == true){
    //for(int i = 0; i < 9; i++){
        display_board();
        player_turn();
    }
}